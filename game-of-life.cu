#include <stdio.h>   /* for printf */
#include <stdlib.h>  /* for malloc, srand, rand */
#include <time.h>    /* for time */


/* cuda specific imports */
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>




#ifdef __unix__  // only unix(POSIX) systems can import those libraries (not: will not work in OS X)

#include <getopt.h>  /* for optopt, getopt, etc */
#include <ctype.h>   /* for usleep */
#include <unistd.h>  /* for opterr */

#else

/**
* Importing libraries for argument parsing wont work.
* USE/CAHNGE THOSE VALUES BEFORE COMPILING INSTEAD
*/

#define COLS 1000
#define ROWS COLS
#define SHOULDPRINT 0
#define BLOKS 50
#define THREADS 200
#define MAXITERATIONS 1000

#endif


#define ALIVE_SYMBOL 'O'
#define DEAD_SYMBOL '.'


struct params {
    int Cols;
    int Rows;
    int max_iterations;               // use -1 for unlimited
    int should_print;
    int alive_probability;           // use -1 for default
    int blocks;                      // use -1 for default
    int threads;                     // use -1 for default
} Params;


void parseCommandLineArguments(int argc, char* argv[]);
void initBoardRand(int* board, int alive_prob);
void printBoard(int* board, int width, FILE* fout);

// Auxiliary functions for locating neighboring cells in a (modular) 2D array represented by an 1D array.
__device__ int mod(int a, int b); // Custom MODULUS function because '%' operator may return negative number
__device__ int cell_right(int myid, int _N, int _M, int n);
__device__ int cell_left(int myid, int _N, int _M, int n);
__device__ int cell_down(int myid, int _N, int _M, int n);
__device__ int cell_up(int myid, int _N, int _M, int n);
__device__ int cell_up_right(int myid, int _N, int _M, int n);
__device__ int cell_down_right(int myid, int _N, int _M, int n);
__device__ int cell_down_left(int myid, int _N, int _M, int n);
__device__ int cell_up_left(int myid, int _N, int _M, int n);

__global__ void playGeneration(int *board, int *sums, int *changeHappened, int width, int size);


int main(int argc, char* argv[]) {

    srand((unsigned)time(NULL));

    parseCommandLineArguments(argc, argv);

    int size = (Params.Rows * Params.Cols) * sizeof(int);

    int *board = (int*)malloc(size);

    int *c_board, *c_sums;

    hipMalloc((void**)&c_board, size);
    hipMalloc((void**)&c_sums, size);



    int i, j;
    int stateChanged;

    initBoardRand(board, Params.alive_probability);

    hipMemcpy(c_board, board, size, hipMemcpyHostToDevice);

    if (Params.should_print) printBoard(board, Params.Cols, stdout);

    clock_t start_clock = clock();
    for (i = 1; (Params.max_iterations == -1) ? 1 : i < Params.max_iterations; i++) {

        playGeneration <<<Params.blocks, Params.threads >>>(board, c_sums, &stateChanged, Params.Cols, Params.Cols*Params.Rows);

        if (Params.should_print) {

            hipMemcpy(board, c_board, size, hipMemcpyDeviceToHost);

            for (j = 0; j<50; j++) printf("\n");
            printBoard(board, Params.Cols, stdout);

#ifdef __unix__
            usleep(333 * 1000);
#endif
        }

        if (!stateChanged) {
            break;
        }
    }

    clock_t stop_clock = clock();

    double elapsed_time = (double)(stop_clock - start_clock) / CLOCKS_PER_SEC;
    printf("ELAPSED TIME : %f \n", elapsed_time);

    free(board);

    hipFree(c_board);
    hipFree(c_sums);
}



__global__ void playGeneration(int *board, int *sums, int *changeHappened, int width, int size) {

    int i;

    // -------- Count neighbours and store them in a parallel array ------ //

    int local_size = size / gridDim.x;
    int from = blockIdx.x * blockDim.x + threadIdx.x;
    int to = from + local_size;

    for (i = from; i < to; i++) {

        sums[i] = board[cell_up_left(i, width, width, size)]
                  + board[cell_up(i, width, width, size)]
                  + board[cell_up_right(i, width, width, size)]
                  + board[cell_left(i, width, width, size)]
                  + board[cell_right(i, width, width, size)]
                  + board[cell_down_left(i, width, width, size)]
                  + board[cell_down(i, width, width, size)]
                  + board[cell_down_right(i, width, width, size)];


    }
    // --------------------------------------------------- //

    int was_dead;
    int n_neighbours;
    int change = 0;


    // ------------- Based on the # of neighbours, compute next state for each cell -------- //
    for (i = 0; i < size; i++) {

        was_dead = board[i] == 0;
        n_neighbours = sums[i];


        if (n_neighbours < 2) {
            if (was_dead) {
                change = 0;
            } else {
                change = 1;
                board[i] = 0;
            }
        } else if (n_neighbours == 2) {
            change = 0;
            //board[i] = board[i];

        } else if (n_neighbours == 3) {
            if (was_dead) {
                change = 1;
                board[i] = 1;
            } else {
                change = 0;
            }


        } else if (n_neighbours >= 4 && n_neighbours <= 8) {
            if (was_dead) {
                change = 0;
            } else {
                change = 1;
            }
            board[i] = 0;

        }


        *changeHappened = *changeHappened || change;
    }

}












// -------------- Auxiliary business functions ------------------------//



    void initBoardRand(int* board, int alive_prob) {
        int i;
        for (i = 0; i<Params.Rows*Params.Cols; i++) {

            board[i] = rand() % 2 == 0;

        }
    }

void printBoard(int *board, int width, FILE* fout) {

    int i;
    for (i = 0; i<Params.Rows*Params.Cols; i++) {
        fprintf(fout, "%c ", (board[i] == 1) ? ALIVE_SYMBOL : DEAD_SYMBOL);
        if (i % width == width - 1) {
            fprintf(fout, "\n");
        }

    }
}



void parseCommandLineArguments(int argc, char* argv[]) {


#if __unix__

    int c;
    opterr = 0;

    Params.Cols              = -1;
    Params.Rows              = -1;
    Params.alive_probability = 15;
    Params.max_iterations    = 100;




    static int print_flag = 0;
    static int help_flag = 0;

    static struct option long_options[] =
    {
    {"size",       required_argument, 0,           's'},
    {"alive-prob", required_argument, 0,           'a'},
    {"end",        required_argument, 0,           'e'},
    {"print",      no_argument,       &print_flag, 'p'},
    {"help",       no_argument,       &help_flag,  'h'},
    {"blocks",     required_argument, 0,           'b'},
    {"threads",    required_argument, 0,           't'},
    {0, 0, 0, 0}
    };

    int option_index = 0;

    while ((c = getopt_long(argc, argv, "s:a:e:b:t:ph", long_options, &option_index)) != -1)
    switch (c)
    {
    case 's':
    Params.Cols = atoi(optarg);
    Params.Rows = atoi(optarg);

    break;


    case 'a':
    Params.alive_probability = atoi(optarg);
    break;

    case 'e':
    Params.max_iterations = atoi(optarg);
    break;

    case 'b':
    Params.blocks = atoi(optarg);
    break;

    case 't':
    Params.blocks = atoi(optarg);
    break;

    case 'p':
    break;

    case 'h':
    break;

    case '?':
    if (optopt == 'e' || optopt == 'a' || optopt == 't' || optopt == 's' || optopt == 'b' || optopt == 't')
    fprintf (stderr, "Option -%c requires an argument.\n", optopt);

    else if (isprint (optopt))
    fprintf (stderr, "Unknown option `-%c'.\n", optopt);
    else
    fprintf (stderr, "Unknown option character `\\x%x'.\n", optopt);

    exit(1);
    break;
    default:
    break;
    }


    if (help_flag){
    char helpMessage[] = "Usage: game -s SIZE [OPTION]...\n\nA parallel implementation of Game Of Life using Cuda.\n\n\nMANDATORY OPTIONS:\n\n  -s, --size SIZE         Use board of SIZE rows and SIZE columns.\n\nIn this version SIZE must be devided by the square root of the number of MPI processes.\n\nOPTIONAL OPTIONS:\n\n  -b, --blocks BLOCKS     Execute with BLOCKS Cuda blocks (Use -1 for a predefined number - Default).\n  -t, --threads THR       Execute with THR threads (Use -1 for a predefined number - Default).\n  -a, --alive-prob PRO    Use PRO (in %) as probability of spawning an alive creature at each cell in the initial state. (default 15)\n  -e, --end NGEN          End the game after NGEN generations. (default 100) (Use -1 for infinite)\n  -p, --print             Print each state on screen.\n  -h, --help              Display this message and exit.\n";

    #pragma GCC diagnostic ignored "-Wformat-security"
    printf(helpMessage);
    exit(0);
    }

    if (Params.Rows == -1 || Params.Cols == -1){
    fprintf(stderr, "Option -s is required.\n");
    exit(3);
    }

    if (Params.blocks == -1){
    Params.blocks = 8;
    }

    if (Params.threads == -1){
    Params.threads = 100;
    }

    Params.should_print = print_flag;

#else
    Params.Cols = COLS;
    Params.Rows = ROWS;
    Params.blocks = BLOCKS;
    Params.threads = THREADS;
    Params.max_iterations = MAXITERATIONS;
    Params.should_print = SHOULDPRINT;
    Params.alive_probability = 15;
#endif
}




__device__ int mod(int a, int b) {
    int ret = a%b;
    if (ret<0) {
        ret += b;
    }
    return ret;

}

// Auxiliary functions for locating neighbouring cells
__device__ int cell_right(int myid, int _N, int _M, int n) {
    return mod((myid + 1), _M) + (myid / _M) * _M;
}

__device__ int cell_left(int myid, int _N, int _M, int n) {
    return (myid / _M) * _M + mod((myid - 1), _M);
}

__device__ int cell_down(int myid, int _N, int _M, int n) {
    return mod(myid, _M) + mod((myid / _M + 1), _N) * _M;
}

__device__ int cell_up(int myid, int _N, int _M, int n) {
    return mod(myid, _M) + mod((myid / _M - 1), _N) * _M;
}

__device__ int cell_up_right(int myid, int _N, int _M, int n) {
    return cell_up(cell_right(myid, _N, _M, n), _N, _M, n);
}
__device__ int cell_down_right(int myid, int _N, int _M, int n) {
    return cell_down(cell_right(myid, _N, _M, n), _N, _M, n);
}

__device__ int cell_down_left(int myid, int _N, int _M, int n) {
    return cell_down(cell_left(myid, _N, _M, n), _N, _M, n);
}

__device__ int cell_up_left(int myid, int _N, int _M, int n) {
    return cell_up(cell_left(myid, _N, _M, n), _N, _M, n);
}







